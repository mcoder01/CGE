#include "hip/hip_runtime.h"
#include "../headers/math_util.h"

/**
 * Computes a matrix-to-matrix multiplication (rows by columns) through matrix d_A (mxk) 
 * and matrix d_B (kxn) through the cuBLAS library. The result is stored into the matrix d_C (mxn).
 */
void matrixMatrixMultiply(double* d_A, double* d_B, double* d_C, int m, int k, int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double alpha = 1, beta = 0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    hipblasDestroy(handle);
}

/**
 * Parallelize the sum of the rows of A with the vector v.
 */
__global__ void add(double* A, double* v, int m, int n, double* result) {
    int start, size;
    distribute(m, 1, &start, &size);
    for (int i = start; i < start+size; i++)
        if (i < m)
            for (int j = 0; j < n; j++) {
                int idx = i*n+j;
                result[idx] = A[idx]+v[j];
            }
}

/**
 * Parallelize the subtraction between the rows of A and the vector v.
 */
__global__ void sub(double* A, double* v, int m, int n, double* result) {
    int start, size;
    distribute(m, 1, &start, &size);
    for (int i = start; i < start+size; i++)
        if (i < m)
            for (int j = 0; j < n; j++) {
                int idx = i*n+j;
                result[idx] = A[idx]-v[j];
            }
}

/**
 * Generates the Euler rotation matrix of the given angle and for the given axis.
 * The result is stored into `matrix`.
 */
__global__ void rotationMatrix(double* angles, int axis, double* matrix) {
    double sinAngle = sin(angles[axis]), cosAngle = cos(angles[axis]);
    if (axis == 0) {
        matrix[0] = 1;
        matrix[4] = matrix[8] = cosAngle;
        matrix[5] = -sinAngle;
        matrix[7] = sinAngle;
    } else {
        matrix[0] = matrix[8] = cosAngle;
        matrix[2] = sinAngle;
        matrix[4] = 1;
        matrix[6] = -sinAngle;
    }
}

/**
 * Rotates the given point by the given angles for X and Y axes.
 * The rotation is firstly performed on the Y-axis and then on the X-axis.
 */
void rotate(double* d_points, double* angles, int num_points, double* output) {
    double *rotX, *rotY;
    hipMalloc((void**) &rotX, 9*sizeof(double));
    hipMemset(rotX, 0, 9*sizeof(double));
    hipMalloc((void**) &rotY, 9*sizeof(double));
    hipMemset(rotY, 0, 9*sizeof(double));

    rotationMatrix<<<1,1>>>(angles, 1, rotY);
    matrixMatrixMultiply(rotY, d_points, output, 3, 3, num_points);
    rotationMatrix<<<1,1>>>(angles, 0, rotX);
    matrixMatrixMultiply(rotX, output, output, 3, 3, num_points);

    hipFree(rotX);
    hipFree(rotY);
}