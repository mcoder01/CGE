#include "../headers/cuda_util.h"

/**
 * Uploads an Obj3d object on the GPU
 */
DeviceObj3d uploadObj3dToDevice(Obj3d obj) {
    DeviceObj3d d_obj;
    hipMalloc((void**) &d_obj.pos, 3*sizeof(double));
    hipMemcpy(d_obj.pos, obj.pos, 3*sizeof(double), hipMemcpyDefault);
    hipMalloc((void**) &d_obj.rot, 3*sizeof(double));
    hipMemcpy(d_obj.rot, obj.rot, 3*sizeof(double), hipMemcpyDefault);
    return d_obj;
}

/**
 * Uploads a Model object to the GPU
 */
DeviceModel uploadModelToDevice(Model model) {
    DeviceModel data;
    hipMalloc((void**) &data.points, model.points.size()*sizeof(double));
    hipMalloc((void**) &data.texCoords, model.texCoords.size()*sizeof(double));
    hipMalloc((void**) &data.vertices, model.vertices.size()*sizeof(int));
    hipMalloc((void**) &data.texVertices, model.texVertices.size()*sizeof(int));

    hipMemcpy(data.points, model.points.data(), model.points.size()*sizeof(double), hipMemcpyDefault);
    hipMemcpy(data.texCoords, model.texCoords.data(), model.texCoords.size()*sizeof(double), hipMemcpyDefault);
    hipMemcpy(data.vertices, model.vertices.data(), model.vertices.size()*sizeof(int), hipMemcpyDefault);
    hipMemcpy(data.texVertices, model.texVertices.data(), model.texVertices.size()*sizeof(int), hipMemcpyDefault);

    data.points_size = model.points.size();
    data.coords_size = model.texCoords.size();
    data.vertices_size = model.vertices.size();
    return data;
}

/**
 * Uploads the pixels of the given texture on the GPU
 */
Texture uploadTextureToDevice(Texture texture) {
    Texture data = texture;
    int len = texture.width*texture.height;
    hipMalloc((void**) &data.pixels, len*sizeof(int));
    hipMemcpy(data.pixels, texture.pixels, len*sizeof(int), hipMemcpyDefault);
    return data;
}

/**
 * Uploads mesh data to the GPU
 */
DeviceMesh uploadMeshToDevice(Mesh mesh) {
    DeviceObj3d obj = uploadObj3dToDevice(mesh);
    return {
        obj.pos, obj.rot,
        uploadModelToDevice(mesh.model),
        uploadTextureToDevice(mesh.texture)
    };
}

/**
 * Allocates the array of pixels and the z-buffer on the GPU
 */
DeviceScreen allocateScreenDataOnDevice(DeviceScreen screen) {
    int len = screen.width*screen.height;
    hipMalloc((void**) &screen.pixels, len*sizeof(int));
    hipMalloc((void**) &screen.zbuffer, len*sizeof(double));
    return screen;
}

/**
 * Deallocates the Obj3d on the GPU
 */
void deleteDeviceObj3d(DeviceObj3d obj) {
    hipFree(obj.pos);
    hipFree(obj.rot);
}

/**
 * Deallocates the given model on the GPU
 */
void deleteDeviceModel(DeviceModel model) {
    hipFree(model.points);
    hipFree(model.texCoords);
    hipFree(model.vertices);
    hipFree(model.texVertices);
}

/**
 * Deallocates pixels of the texture on the GPU
 */
void deleteDeviceTexture(Texture texture) {
    hipFree(texture.pixels);
}

/**
 * Deallocates mesh data on the GPU
 */
void deleteDeviceMesh(DeviceMesh mesh) {
    deleteDeviceObj3d(mesh);
    deleteDeviceModel(mesh.model);
    deleteDeviceTexture(mesh.texture);
}

/**
 * Updates the visualized pixels with the ones computed on the GPU
 */
void downloadDevicePixels(DeviceScreen screen, int* pixels) {
    int len = screen.width*screen.height;
    hipMemcpy(pixels, screen.pixels, len*sizeof(int), hipMemcpyDefault);
}