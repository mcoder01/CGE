#include "hip/hip_runtime.h"
#include "../headers/world.h"
#include "../headers/math_util.h"
#include "../headers/render.h"

/**
 * Performs the dot product between two vectors.
 */
__host__ __device__ double dotProduct(double* a, double* b, int dim) {
    double sum = 0;
    for (int i = 0; i < dim; i++)
        sum += a[i]*b[i];
    return sum;
}

/**
 * Performs the normalization of the given vector in-place.
 */
__host__ __device__ void normalize(double* v, int len) {
    double mag = sqrt(dotProduct(v, v, len));
    for (int i = 0; i < len; i++)
        v[i] /= mag;
}

/**
 * Check if the given vertex is in the frustum
 */
__host__ __device__ bool isInFrustum(Vertex v, Plane plane) {
    double pd = dotProduct(plane.position, plane.normal, 3);
    double vd = dotProduct(v.position, plane.normal, 3);
    return vd-pd >= 0;
}

/**
 * Returns the factor which describes the intersection point between 
 * a plane and a segment represented by its ends (vectors `a` and `b`).
 * Input vectors must be of three dimensions.
 */
__host__ __device__ double frustumIntersection(double* a, double* b, Plane plane) {
    double pd = dotProduct(plane.position, plane.normal, 3);
    double ad = dotProduct(a, plane.normal, 3);
    double bd = dotProduct(b, plane.normal, 3);
    return (pd-ad)/(bd-ad);
}

#if DEVICE == 0

World::World(DeviceScreen screen):screen(screen) {}

void World::addMesh(Mesh mesh) {
    objects.push_back(mesh);
}

void World::tick(double deltaTime) {
    objects[0].rot[1] += 0.1*deltaTime;
}

/**
 * Rotates and moves the points of the given mesh in the 3D space with respect
 * to the position and the rotation of the camera and the mesh object.
 */
double* viewMeshPoints(Mesh& mesh, Obj3d& camera) {
    int num_points = mesh.model.points.size()/3;
    double* viewed = new double[mesh.model.points.size()];

    // Rotates mesh points respect to its rotation
    rotate(mesh.model.points.data(), mesh.rot, num_points, viewed);

    // Moves mesh points according to its position and the position of the camera
    double shift[3];
    sub(mesh.pos, camera.pos, 1, 3, shift);
    add(viewed, shift, num_points, 3, viewed);

    // Rotates mesh points with respect to rotation of the camera
    rotate(viewed, camera.rot, num_points, viewed);
    return viewed;
}

/**
 * Computes the normals of the viewed points according to the triangles which compose the mesh.
 * Returns a normal vector for each point of the mesh.
 */
double* computeNormals(Mesh& mesh, double* points) {
    double* normals = new double[mesh.model.points.size()]();
    std::vector<int>& indices = mesh.model.vertices;
    for (int i = 0; i < indices.size(); i += 3) {
        int pidx[3] = {indices[i]*3, indices[i+1]*3, indices[i+2]*3};
        double v1[3], v2[3];
        for (int j = 0; j < 3; j++) {
            v1[j] = points[pidx[1]+j]-points[pidx[0]+j];
            v2[j] = points[pidx[2]+j]-points[pidx[0]+j];
        }

        // Perform the cross product between two sides of the triangle
        double normal[3] = {
            v1[1]*v2[2]-v1[2]*v2[1],
            v1[2]*v2[0]-v1[0]*v2[2],
            v1[0]*v2[1]-v1[1]*v2[0]
        };

        /* Normalize the computed normal and sum it to the normals 
           related to the points involved into the computation. */
        normalize(normal, 3);
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++)
                normals[pidx[j]+k] += normal[k];
    }

    return normals;
}

/**
 * Takes the processed points, UV and normals and generates an array of vertices which represent the mesh triangles.
 * For optimization purposes, the method automatically detects invisible triangles and ignore them during the generation of the vertices.
 */
int decomposeFaces(Mesh& mesh, double* points, double* texCoords, double* normals, Vertex* vertices) {
    std::vector<int>& indices = mesh.model.vertices;
    std::vector<int>& texIndices = mesh.model.texVertices;
    int faces = 0;
    for (int i = 0; i < indices.size(); i += 3) {
        int face = -1;
        for (int j = 0; j < 3; j++) {
            int pointIdx = indices[i+j], texIdx = texIndices[i+j];
            normalize(normals+pointIdx*3, 3);
            if (face >= 0) {
                int idx = face*3+j;
                memcpy(vertices[idx].position, points+pointIdx*3, 3*sizeof(double));
                memcpy(vertices[idx].texCoord, texCoords+texIdx*2, 2*sizeof(double));
                memcpy(vertices[idx].normal, normals+pointIdx*3, 3*sizeof(double));
            } else if (dotProduct(points+pointIdx*3, normals+pointIdx*3, 3) < 0) {
                face = faces++;
                j = -1;
            }
        }
    }

    return faces;
}

/**
 * Clips each triangle against the given plane in the 3D space. The clipped triangles
 * are stored into a the array of vertices `clipped`, while its length is stored into `len`.
 */
void clipFaces(Vertex* vertices, int faces, Plane plane, Vertex* clipped, int* len) {
    *len = 0;
    for (int i = 0; i < faces; i++) {
        // For each triangle, divide points inside the frustum from outside ones
        Vertex inside[3], outside[3];
        int in_num = 0, out_num = 0;
        for (int j = 0; j < 3; j++) {
            Vertex v = vertices[i*3+j];
            if (isInFrustum(v, plane))
                inside[in_num++] = v;
            else outside[out_num++] = v;
        }

        // Clip the triangle relying on the number of inside and outside points
        if (in_num == 1) {
            int index = ((*len)++)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[0].position, outside[1].position, plane);
            clipped[index] = inside[0];
            clipped[index+1] = lerpVertex(&inside[0], &outside[0], t1);
            clipped[index+2] = lerpVertex(&inside[0], &outside[1], t2);
        } else if (in_num == 2) {
            int idx1 = ((*len)++)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[1].position, outside[0].position, plane);

            clipped[idx1] = inside[0];
            clipped[idx1+1] = inside[1];
            clipped[idx1+2] = lerpVertex(&inside[0], &outside[0], t1);

            int idx2 = ((*len)++)*3;
            clipped[idx2] = inside[1];
            clipped[idx2+1] = clipped[idx1+2];
            clipped[idx2+2] = lerpVertex(&inside[1], &outside[0], t2);
        } else if (in_num == 3) {
            // The triangle should not be clipped
            int index = ((*len)++)*3;
            memcpy(clipped+index, inside, 3*sizeof(Vertex));
        }
    }
}

/**
 * Clips the given triangles against the near and far planes of the frustum.
 */
Vertex* clipFaces(Vertex* vertices, int& faces) {
    Vertex nearClipped[faces*6];
    clipFaces(vertices, faces, {0, 0, -1, 0, 0, -1}, nearClipped, &faces);
    Vertex* clipped = new Vertex[faces*6];
    clipFaces(nearClipped, faces, {0, 0, -100, 0, 0, 1}, clipped, &faces);
    return clipped;
}

/**
 * Perform the perspective projection of the given vertices.
 */
void project(Vertex* vertices, int num_points, DeviceScreen screen) {
    for (int i = 0; i < num_points; i++) {
        double absZ = abs(vertices[i].position[2]);
        vertices[i].projection[0] = vertices[i].position[0]/absZ*screen.scale+screen.width/2;
        vertices[i].projection[1] = -vertices[i].position[1]/absZ*screen.scale+screen.height/2;
    }
}

/**
 * Returns the rectangle which describe the smallest area containing the projections
 * of all the vertices
 */
SDL_Rect computeRenderArea(Vertex* vertices, int num_points, DeviceScreen screen) {
    int diagonal[4] = {INT_MAX, INT_MAX, 0, 0};
    for (int i = 0; i < num_points; i++) {
        int x = min(max((int) vertices[i].projection[0], 0), screen.width-1);
        int y = min(max((int) vertices[i].projection[1], 0), screen.height-1);

        diagonal[0] = min(diagonal[0], x);
        diagonal[1] = min(diagonal[1], y);
        diagonal[2] = max(diagonal[2], x);
        diagonal[3] = max(diagonal[3], y);
    }

    return {diagonal[0], diagonal[1], diagonal[2]-diagonal[0], diagonal[3]-diagonal[1]};
}

/**
 * This method is called each frame to process and render all the meshes of the scene.
 */
void World::drawObjects(SDL_Surface* surface, Obj3d camera) {
    screen.pixels = (int*) surface->pixels;
    screen.zbuffer = new double[screen.width*screen.height]();

    SDL_Rect screenRect = {0, 0, screen.width, screen.height};
    SDL_FillRect(surface, &screenRect, 0x000000);

    for (Mesh mesh : objects) {
        // Compute all the data required to render the mesh
        double* points = viewMeshPoints(mesh, camera);
        double* normals = computeNormals(mesh, points);
        
        // Decompose points into triangles and clip them
        Vertex vertices[mesh.model.vertices.size()];
        int faces = decomposeFaces(mesh, points, mesh.model.texCoords.data(), normals, vertices);
        Vertex* clipped = clipFaces(vertices, faces);

        /* Project the final vertices, compute the rendering area 
         * of the screen and fit the texture on the mesh */
        project(clipped, faces*3, screen);
        SDL_Rect area = computeRenderArea(clipped, faces*3, screen);
        raster(clipped, faces*3, mesh.texture, screen, area);

        // Free all the memory allocated on the GPU for this mesh
        delete[] points;
        delete[] normals;
        delete[] clipped;
    }

    delete[] screen.zbuffer;
}

#elif DEVICE == 1

World::World(DeviceScreen screen) {
    this->screen = allocateScreenDataOnDevice(screen);
}

void World::addMesh(Mesh mesh) {
    objects.push_back(mesh);
}

void World::tick(double deltaTime) {
    //objects[0].rot[1] += 0.1*deltaTime;
}

/**
 * Rotates and moves the points of the given mesh in the 3D space with respect
 * to the position and the rotation of the camera and the mesh object.
 */
double* viewMeshPoints(DeviceMesh& mesh, DeviceObj3d camera) {
    int num_points = mesh.model.points_size/3;
    double* viewed;

    // Rotates mesh points respect to its rotation
    hipMalloc((void**) &viewed, mesh.model.points_size*sizeof(double));
    rotate(mesh.model.points, mesh.rot, num_points, viewed);

    // Moves mesh points according to its position and the position of the camera
    double* shift;
    hipMalloc((void**) &shift, 3*sizeof(double));
    sub<<<3,1>>>(mesh.pos, camera.pos, 1, 3, shift);
    add<<<48,64>>>(viewed, shift, num_points, 3, viewed);
    hipFree(shift);

    // Rotates mesh points with respect to rotation of the camera
    rotate(viewed, camera.rot, num_points, viewed);
    return viewed;
}

/**
 * Transfers the UV coordinates of the texture on the GPU
 */
double* uploadTexCoordsToDevice(DeviceMesh& mesh) {
    double* texCoords;
    hipMalloc((void**) &texCoords, mesh.model.coords_size*sizeof(double));
    hipMemcpy(texCoords, mesh.model.texCoords, mesh.model.coords_size*sizeof(double), hipMemcpyDefault);
    return texCoords;
}

/**
 * Computes the normals of the viewed points according to the triangles which compose the mesh.
 * Returns a normal vector for each point of the mesh.
 */
__global__ void computeNormals(DeviceMesh& mesh, double* points, double* normals) {
    int start, size;
    distribute(mesh.model.vertices_size, 3, &start, &size);

    int* indices = mesh.model.vertices;
    for (int i = start; i < start+size; i += 3) {
        if (i >= mesh.model.vertices_size)
            return;

        int pidx[3] = {indices[i]*3, indices[i+1]*3, indices[i+2]*3};
        double v1[3], v2[3];
        for (int j = 0; j < 3; j++) {
            v1[j] = points[pidx[1]+j]-points[pidx[0]+j];
            v2[j] = points[pidx[2]+j]-points[pidx[0]+j];
        }

        // Perform the cross product between two sides of the triangle
        double normal[3] = {
            v1[1]*v2[2]-v1[2]*v2[1],
            v1[2]*v2[0]-v1[0]*v2[2],
            v1[0]*v2[1]-v1[1]*v2[0]
        };

        /* Normalize the computed normal and sum it to the normals 
           related to the points involved into the computation. */
        normalize(normal, 3);
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++)
                atomicAdd(normals+pidx[j]+k, normal[k]);
    }
}

double* computeNormals(DeviceMesh& mesh, double* points) {
    double* normals;
    hipMalloc((void**) &normals, mesh.model.points_size*sizeof(double));
    hipMemset(normals, 0, mesh.model.points_size*sizeof(double));
    computeNormals<<<48,64>>>(mesh, points, normals);
    return normals;
}

/**
 * Takes the processed points, UV and normals and generates an array of vertices which represent the mesh triangles.
 * For optimization purposes, the method automatically detects invisible triangles and ignore them during the generation of the vertices.
 */
__global__ void decomposeFaces(DeviceMesh mesh, double* points, double* texCoords, double* normals, Vertex* vertices, int* len) {
    int start, size;
    distribute(mesh.model.vertices_size, 3, &start, &size);
    for (int i = start; i < start+size; i += 3) {
        if (i >= mesh.model.vertices_size)
            return;

        int face = -1;
        for (int j = 0; j < 3; j++) {
            int pointIdx = mesh.model.vertices[i+j], texIdx = mesh.model.texVertices[i+j];
            normalize(normals+pointIdx*3, 3);
            if (face >= 0) {
                int idx = face*3+j;
                memcpy(vertices[idx].position, points+pointIdx*3, 3*sizeof(double));
                memcpy(vertices[idx].texCoord, texCoords+texIdx*2, 2*sizeof(double));
                memcpy(vertices[idx].normal, normals+pointIdx*3, 3*sizeof(double));
            } else if (dotProduct(points+pointIdx*3, normals+pointIdx*3, 3) < 0) {
                face = atomicAdd(len, 1);
                j = -1;
            }
        }
    }
}

Vertex* decomposeFaces(DeviceMesh mesh, double* points, double* texCoords, double* normals, int& size) {
    Vertex* vertices;
    int* d_size;
    hipMalloc((void**) &vertices, mesh.model.vertices_size*sizeof(Vertex));
    hipMalloc((void**) &d_size, sizeof(int));
    hipMemset(d_size, 0, sizeof(int));
    decomposeFaces<<<48,64>>>(mesh, points, texCoords, normals, vertices, d_size);
    hipMemcpy(&size, d_size, sizeof(int), hipMemcpyDefault);
    hipFree(d_size);
    return vertices;
}

/**
 * Clips each triangle against the given plane in the 3D space. The clipped triangles
 * are stored into a the array of vertices `clipped`, while its length is stored into `len`.
 */
__global__ void clipFaces(Vertex* vertices, int faces, Plane plane, Vertex* clipped, int* len) {
    int start, size;
    distribute(faces, 1, &start, &size);
    for (int i = start; i < start+size; i++) {
        if (i >= faces) break;

        // For each triangle, divide points inside the frustum from outside ones
        Vertex inside[3], outside[3];
        int in_num = 0, out_num = 0;
        for (int j = 0; j < 3; j++) {
            Vertex v = vertices[i*3+j];
            if (isInFrustum(v, plane))
                inside[in_num++] = v;
            else outside[out_num++] = v;
        }

        // Clip the triangle relying on the number of inside and outside points
        if (in_num == 1) {
            int index = atomicAdd(len, 1)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[0].position, outside[1].position, plane);
            clipped[index] = inside[0];
            clipped[index+1] = lerpVertex(&inside[0], &outside[0], t1);
            clipped[index+2] = lerpVertex(&inside[0], &outside[1], t2);
        } else if (in_num == 2) {
            int idx1 = atomicAdd(len, 1)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[1].position, outside[0].position, plane);

            clipped[idx1] = inside[0];
            clipped[idx1+1] = inside[1];
            clipped[idx1+2] = lerpVertex(&inside[0], &outside[0], t1);

            int idx2 = atomicAdd(len, 1)*3;
            clipped[idx2] = inside[1];
            clipped[idx2+1] = clipped[idx1+2];
            clipped[idx2+2] = lerpVertex(&inside[1], &outside[0], t2);
        } else if (in_num == 3) {
            // The triangle should not be clipped
            int index = atomicAdd(len, 1)*3;
            memcpy(clipped+index, inside, 3*sizeof(Vertex));
        }
    }
}

Vertex* clipFacesAgainstPlane(Vertex* vertices, int& faces, Plane plane) {
    Vertex* clipped;
    int* output_size;
    hipMalloc((void**) &clipped, 6*faces*sizeof(Vertex));
    hipMalloc((void**) &output_size, sizeof(int));
    hipMemset(output_size, 0, sizeof(int));
    clipFaces<<<48,64>>>(vertices, faces, plane, clipped, output_size);
    hipMemcpy(&faces, output_size, sizeof(int), hipMemcpyDefault);
    hipFree(output_size);
    return clipped;
}

/**
 * Clips the given triangles against the near and far planes of the frustum.
 */
Vertex* clipFaces(Vertex* vertices, int& faces) {
    Vertex* nearClipped = clipFacesAgainstPlane(vertices, faces, {0, 0, -1, 0, 0, -1});
    Vertex* clipped = clipFacesAgainstPlane(nearClipped, faces, {0, 0, -100, 0, 0, 1});
    hipFree(nearClipped);
    return clipped;
}

/**
 * Perform the perspective projection of the given vertices.
 */
__global__ void project(Vertex* vertices, int num_points, DeviceScreen screen) {
    int start, size;
    distribute(num_points, 1, &start, &size);
    for (int i = start; i < start+size; i++) {
        if (i >= num_points)
            return;

        double absZ = abs(vertices[i].position[2]);
        vertices[i].projection[0] = vertices[i].position[0]/absZ*screen.scale+screen.width/2;
        vertices[i].projection[1] = -vertices[i].position[1]/absZ*screen.scale+screen.height/2;
    }
}

/**
 * This method finds the smallest diagonal (described by the top left and bottom right corners) of
 * the rectangle which contains the projections of all the given vertices.
 */
__global__ void findDiagonal(Vertex* vertices, int num_points, DeviceScreen screen, int* diagonal) {
    int start, size;
    distribute(num_points, 1, &start, &size);

    int minX = INT_MAX, minY = INT_MAX;
    int maxX = 0, maxY = 0;
    for (int i = start; i < start+size; i++) {
        if (i >= num_points)
            break;

        int x = min(max((int) vertices[i].projection[0], 0), screen.width-1);
        int y = min(max((int) vertices[i].projection[1], 0), screen.height-1);

        minX = min(minX, x);
        maxX = max(maxX, x);
        minY = min(minY, y);
        maxY = max(maxY, y);
    }

    if (start < num_points) {
        atomicMin(&diagonal[0], minX);
        atomicMin(&diagonal[1], minY);
        atomicMax(&diagonal[2], maxX);
        atomicMax(&diagonal[3], maxY);
    }
}

/**
 * Returns the rectangle which describe the smallest area containing the projections
 * of all the vertices
 */
SDL_Rect computeRenderArea(Vertex* vertices, int num_points, DeviceScreen screen) {
    int diagonal[4] = {INT_MAX, INT_MAX, 0, 0};
    int* d_diagonal;
    hipMalloc((void**) &d_diagonal, sizeof(diagonal));
    hipMemcpy(d_diagonal, diagonal, sizeof(diagonal), hipMemcpyDefault);
    findDiagonal<<<48,64>>>(vertices, num_points, screen, d_diagonal);
    hipMemcpy(diagonal, d_diagonal, sizeof(diagonal), hipMemcpyDefault);
    hipFree(d_diagonal);
    return {diagonal[0], diagonal[1], diagonal[2]-diagonal[0]+1, diagonal[3]-diagonal[1]+1};
}

/**
 * This method is called each frame to process and render all the meshes of the scene.
 */
void World::drawObjects(SDL_Surface* surface, Obj3d camera) {
    initScreen<<<8,224>>>(screen); // Clean the screen

    DeviceObj3d d_camera = uploadObj3dToDevice(camera);
    for (Mesh mesh : objects) {
        // Compute all the data required to render the mesh
        DeviceMesh d_mesh = uploadMeshToDevice(mesh);
        double* points = viewMeshPoints(d_mesh, d_camera);
        double* texCoords = uploadTexCoordsToDevice(d_mesh);
        double* normals = computeNormals(d_mesh, points);
        
        // Decompose points into triangles and clip them
        int faces;
        Vertex* vertices = decomposeFaces(d_mesh, points, texCoords, normals, faces);
        Vertex* clipped = clipFaces(vertices, faces);
        hipFree(vertices);

        /* Project the final vertices, compute the rendering area 
         * of the screen and fit the texture on the mesh */
        project<<<48,64>>>(clipped, faces*3, screen);
        SDL_Rect area = computeRenderArea(clipped, faces*3, screen);
        raster<<<48,64>>>(clipped, faces*3, d_mesh.texture, screen, area);

        // Free all the memory allocated on the GPU for this mesh
        deleteDeviceMesh(d_mesh);
        hipFree(points);
        hipFree(texCoords);
        hipFree(normals);
        hipFree(clipped);
    }

    deleteDeviceObj3d(d_camera);
    downloadDevicePixels(screen, (int*) surface->pixels); // Update pixels
}

#endif