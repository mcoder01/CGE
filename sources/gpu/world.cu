#include "hip/hip_runtime.h"
World::World(DeviceScreen screen) {
    this->screen = allocateScreenDataOnDevice(screen);
}

/**
 * Rotates and moves the points of the given mesh in the 3D space with respect
 * to the position and the rotation of the camera and the mesh object.
 */
double* viewMeshPoints(DeviceMesh& mesh, DeviceObj3d camera) {
    int num_points = mesh.model.points_size/3;
    double* viewed;

    // Rotates mesh points respect to its rotation
    hipMalloc((void**) &viewed, mesh.model.points_size*sizeof(double));
    rotate(mesh.model.points, mesh.rot, num_points, viewed);

    // Moves mesh points according to its position and the position of the camera
    double* shift;
    hipMalloc((void**) &shift, 3*sizeof(double));
    sub<<<3,1>>>(mesh.pos, camera.pos, 1, 3, shift);
    add<<<48,64>>>(viewed, shift, num_points, 3, viewed);
    hipFree(shift);

    // Rotates mesh points with respect to rotation of the camera
    rotate(viewed, camera.rot, num_points, viewed);
    return viewed;
}

/**
 * Transfers the UV coordinates of the texture on the GPU
 */
double* uploadTexCoordsToDevice(DeviceMesh& mesh) {
    double* texCoords;
    hipMalloc((void**) &texCoords, mesh.model.coords_size*sizeof(double));
    hipMemcpy(texCoords, mesh.model.texCoords, mesh.model.coords_size*sizeof(double), hipMemcpyDefault);
    return texCoords;
}

__device__ void computeNormal(double* points, int* indices, double* normal) {
    int pidx[3] = {indices[0]*3, indices[1]*3, indices[2]*3};
    double v1[3], v2[3];
    for (int i = 0; i < 3; i++) {
        v1[i] = points[pidx[1]+i]-points[pidx[0]+i];
        v2[i] = points[pidx[2]+i]-points[pidx[0]+i];
    }

    // Perform the cross product between two sides of the triangle
    normal[0] = v1[1]*v2[2]-v1[2]*v2[1];
    normal[1] = v1[2]*v2[0]-v1[0]*v2[2];
    normal[2] = v1[0]*v2[1]-v1[1]*v2[0];
    normalize(normal, 3);
}

/**
 * Computes smooth normals of the viewed points according to the triangles which compose the mesh.
 * Returns a normal vector for each point of the mesh.
 */
__global__ void computeSmoothNormals(DeviceMesh& mesh, double* points, double* normals) {
    int start, size;
    distribute(mesh.model.vertices_size, 3, &start, &size);

    int* indices = mesh.model.vertices;
    for (int i = start; i < start+size; i += 3) {
        if (i >= mesh.model.vertices_size)
            return;

        double normal[3];
        computeNormal(points, indices+i, normal);
        for (int j = 0; j < 3; j++)
            for (int k = 0; k < 3; k++)
                atomicAdd(normals+indices[i+j]+k, normal[k]);
    }
}

double* computeSmoothNormals(DeviceMesh& mesh, double* points) {
    double* normals;
    hipMalloc((void**) &normals, mesh.model.points_size*sizeof(double));
    hipMemset(normals, 0, mesh.model.points_size*sizeof(double));
    computeSmoothNormals<<<48,64>>>(mesh, points, normals);
    return normals;
}

/**
 * Takes the processed points, UV and normals and generates an array of vertices which represent the mesh triangles.
 * For optimization purposes, the method automatically detects invisible triangles and ignore them during the generation of the vertices.
 */
__global__ void decomposeFaces(DeviceMesh mesh, double* points, double* texCoords, double* normals, Vertex* vertices, int* len) {
    int start, size;
    distribute(mesh.model.vertices_size, 3, &start, &size);
    for (int i = start; i < start+size; i += 3) {
        if (i >= mesh.model.vertices_size)
            return;

        int face = -1;
        double normal[3];
        computeNormal(points, mesh.model.vertices+i, normal);
        for (int j = 0; j < 3; j++) {
            int pointIdx = mesh.model.vertices[i+j], texIdx = mesh.model.texVertices[i+j];
            if (face >= 0) {
                int idx = face*3+j;
                memcpy(vertices[idx].position, points+pointIdx*3, 3*sizeof(double));
                memcpy(vertices[idx].texCoord, texCoords+texIdx*2, 2*sizeof(double));
                memcpy(vertices[idx].normal, normals+pointIdx*3, 3*sizeof(double));
            } else if (dotProduct(points+pointIdx*3, normal, 3) < 0) {
                face = atomicAdd(len, 1);
                j = -1;
            }
        }
    }
}

Vertex* decomposeFaces(DeviceMesh mesh, double* points, double* texCoords, double* normals, int& size) {
    Vertex* vertices;
    int* d_size;
    hipMalloc((void**) &vertices, mesh.model.vertices_size*sizeof(Vertex));
    hipMalloc((void**) &d_size, sizeof(int));
    hipMemset(d_size, 0, sizeof(int));
    decomposeFaces<<<48,64>>>(mesh, points, texCoords, normals, vertices, d_size);
    hipMemcpy(&size, d_size, sizeof(int), hipMemcpyDefault);
    hipFree(d_size);
    return vertices;
}

/**
 * Compute the culling of the vertices against the given plane of the frustum.
 * Triangles which crosses it are clipped and stored into the
 * array of vertices `clipped`, while its length is stored into `len`.
 */
__global__ void culling(Vertex* vertices, int faces, Plane plane, Vertex* clipped, int* len) {
    int start, size;
    distribute(faces, 1, &start, &size);
    for (int i = start; i < start+size; i++) {
        if (i >= faces) break;

        // For each triangle, divide points inside the frustum from outside ones
        Vertex inside[3], outside[3];
        int in_num = 0, out_num = 0;
        for (int j = 0; j < 3; j++) {
            Vertex v = vertices[i*3+j];
            if (isInFrustum(v, plane))
                inside[in_num++] = v;
            else outside[out_num++] = v;
        }

        // Clip the triangle relying on the number of inside and outside points
        if (in_num == 1) {
            int index = atomicAdd(len, 1)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[0].position, outside[1].position, plane);
            clipped[index] = inside[0];
            clipped[index+1] = lerpVertex(&inside[0], &outside[0], t1);
            clipped[index+2] = lerpVertex(&inside[0], &outside[1], t2);
        } else if (in_num == 2) {
            int idx1 = atomicAdd(len, 1)*3;
            double t1 = frustumIntersection(inside[0].position, outside[0].position, plane);
            double t2 = frustumIntersection(inside[1].position, outside[0].position, plane);

            clipped[idx1] = inside[0];
            clipped[idx1+1] = inside[1];
            clipped[idx1+2] = lerpVertex(&inside[0], &outside[0], t1);

            int idx2 = atomicAdd(len, 1)*3;
            clipped[idx2] = inside[1];
            clipped[idx2+1] = clipped[idx1+2];
            clipped[idx2+2] = lerpVertex(&inside[1], &outside[0], t2);
        } else if (in_num == 3) {
            // The triangle should not be clipped
            int index = atomicAdd(len, 1)*3;
            memcpy(clipped+index, inside, 3*sizeof(Vertex));
        }
    }
}

Vertex* culling(Vertex* vertices, int& faces, Plane plane) {
    Vertex* clipped;
    int* output_size;
    hipMalloc((void**) &clipped, 6*faces*sizeof(Vertex));
    hipMalloc((void**) &output_size, sizeof(int));
    hipMemset(output_size, 0, sizeof(int));
    culling<<<48,64>>>(vertices, faces, plane, clipped, output_size);
    hipMemcpy(&faces, output_size, sizeof(int), hipMemcpyDefault);
    hipFree(output_size);
    return clipped;
}

/**
 * Performs the frustum culling against near and far planes of the frustum.
 */
Vertex* frustumCulling(Vertex* vertices, int& faces) {
    Vertex* nearClipped = culling(vertices, faces, {0, 0, -1, 0, 0, -1});
    Vertex* clipped = culling(nearClipped, faces, {0, 0, -100, 0, 0, 1});
    hipFree(nearClipped);
    return clipped;
}

/**
 * Perform the perspective projection of the given vertices.
 */
__global__ void project(Vertex* vertices, int num_points, DeviceScreen screen) {
    int start, size;
    distribute(num_points, 1, &start, &size);
    for (int i = start; i < start+size; i++) {
        if (i >= num_points)
            return;

        double absZ = abs(vertices[i].position[2]);
        vertices[i].projection[0] = vertices[i].position[0]/absZ*screen.scale+screen.width/2;
        vertices[i].projection[1] = -vertices[i].position[1]/absZ*screen.scale+screen.height/2;
    }
}

/**
 * This method finds the smallest diagonal (described by the top left and bottom right corners) of
 * the rectangle which contains the projections of all the given vertices.
 */
__global__ void findDiagonal(Vertex* vertices, int num_points, DeviceScreen screen, int* diagonal) {
    int start, size;
    distribute(num_points, 1, &start, &size);

    int minX = INT_MAX, minY = INT_MAX;
    int maxX = 0, maxY = 0;
    for (int i = start; i < start+size; i++) {
        if (i >= num_points)
            break;

        int x = min(max((int) vertices[i].projection[0], 0), screen.width-1);
        int y = min(max((int) vertices[i].projection[1], 0), screen.height-1);

        minX = min(minX, x);
        maxX = max(maxX, x);
        minY = min(minY, y);
        maxY = max(maxY, y);
    }

    if (start < num_points) {
        atomicMin(&diagonal[0], minX);
        atomicMin(&diagonal[1], minY);
        atomicMax(&diagonal[2], maxX);
        atomicMax(&diagonal[3], maxY);
    }
}

/**
 * Returns the rectangle which describe the smallest area containing the projections
 * of all the vertices
 */
SDL_Rect computeRenderArea(Vertex* vertices, int num_points, DeviceScreen screen) {
    int diagonal[4] = {INT_MAX, INT_MAX, 0, 0};
    int* d_diagonal;
    hipMalloc((void**) &d_diagonal, sizeof(diagonal));
    hipMemcpy(d_diagonal, diagonal, sizeof(diagonal), hipMemcpyDefault);
    findDiagonal<<<48,64>>>(vertices, num_points, screen, d_diagonal);
    hipMemcpy(diagonal, d_diagonal, sizeof(diagonal), hipMemcpyDefault);
    hipFree(d_diagonal);
    return {diagonal[0], diagonal[1], diagonal[2]-diagonal[0]+1, diagonal[3]-diagonal[1]+1};
}

/**
 * This method is called each frame to process and render all the meshes of the scene.
 */
void World::drawObjects(SDL_Surface* surface, Obj3d camera) {
    hipEvent_t frameStart, frameStop;
    float framePassedTime;
    hipEventCreate(&frameStart);
    hipEventCreate(&frameStop);
    hipEventRecord(frameStart, 0);
    initScreen<<<8,224>>>(screen); // Clean the screen

    DeviceObj3d d_camera = uploadObj3dToDevice(camera);
    for (Mesh mesh : objects) {
        // Compute all the data required to render the mesh
        DeviceMesh d_mesh = uploadMeshToDevice(mesh);

        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        double* points = viewMeshPoints(d_mesh, d_camera);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        viewPointTime += time;

        double* texCoords = uploadTexCoordsToDevice(d_mesh);

        hipEventRecord(start, 0);
        double* normals = computeSmoothNormals(d_mesh, points);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        normalComputationTime += time;
        
        // Decompose points into triangles and clip them
        hipEventRecord(start, 0);
        int faces;
        Vertex* vertices = decomposeFaces(d_mesh, points, texCoords, normals, faces);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        faceDecompositionTime += time;

        hipEventRecord(start, 0);
        Vertex* clipped = frustumCulling(vertices, faces);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        faceClippingTime += time;
        hipFree(vertices);

        /* Project the final vertices, compute the rendering area 
         * of the screen and fit the texture on the mesh */
        hipEventRecord(start, 0);
        project<<<48,64>>>(clipped, faces*3, screen);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        projectionTime += time;

        hipEventRecord(start, 0);
        SDL_Rect area = computeRenderArea(clipped, faces*3, screen);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        areaComputationTime += time;

        hipEventRecord(start, 0);
        raster<<<48,64>>>(clipped, faces*3, d_mesh.texture, screen, area);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        rasterTime += time;

        // Free all the memory allocated on the GPU for this mesh
        deleteDeviceMesh(d_mesh);
        hipFree(points);
        hipFree(texCoords);
        hipFree(normals);
        hipFree(clipped);
    }

    deleteDeviceObj3d(d_camera);
    downloadDevicePixels(screen, (int*) surface->pixels); // Update pixels

    hipEventRecord(frameStop, 0);
    hipEventSynchronize(frameStop);
    hipEventElapsedTime(&framePassedTime, frameStart, frameStop);
    frameTime += framePassedTime;
    frames++;
}